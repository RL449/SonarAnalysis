#include "hip/hip_runtime.h"
﻿#include <iostream> // Standard input/output stream operations
#include <string> // Standard string class
#include <filesystem> // File path / directory operations
#include <fstream> // File stream operations
#include <cmath> // Standard math functions
#include <algorithm> // Implements common algorithms
#include <sndfile.h> // Read / write audio files
#include <stdexcept> // Handles exceptions
#include <complex> // Used for FFT / Hilbert transforms
#include <fftw3.h> // FFT computations
#include <cstring> // String operations
#include <thread> // Support multithreading
#include <mutex> // Synchronize threads
#include <cassert> // Debugging
#include <regex> // Parse date / time
#include <ctime> // Output time formatting
#include <iomanip> // Setfill, setw

// Limit thread count to # of cores
#include <queue> // Sequential reading of files for threads
#include <condition_variable> // Thread synchronization
#include <atomic> // Atomic operations for multithreading

// CUDA
#include <hipfft/hipfft.h> // CUDA FFT
#include <thrust/reduce.h> // Reduction operations
#include <thrust/device_ptr.h> // Smart pointers

using namespace std; // Standard namespace
namespace fs = filesystem; // Rename filesystem

// Declare structs

struct SampleRange {
    int startSample; // First sample index
    int endSample; // Last sample index

    // Constructor with default range
    SampleRange(int start = 1, int end = -1) {
        startSample = start;
        endSample = end;
    }
};

struct BandpassFilter {
    double* filteredTimeSeries; // Time domain signal after filtering
    double* amplitudeSpectrum; // Frequency domain amplitude spectrum
    int length; // # of samples

    // Constructor
    BandpassFilter(double* ts, double* spec, int len) : filteredTimeSeries(ts), amplitudeSpectrum(spec), length(len) {}

    // Destructor
    ~BandpassFilter() {
        delete[] filteredTimeSeries;
        delete[] amplitudeSpectrum;
    }
};

struct Correlation {
    double* correlationValues; // Cross-correlation values between two signals
    double* lags; // Corresponding lag values
    int length; // Length of the arrays

    // Constructor
    Correlation(double* corr, double* lag, int len) : correlationValues(corr), lags(lag), length(len) {}

    // Destructor
    ~Correlation() {
        delete[] correlationValues;
        delete[] lags;
    }
};

// Extracted audio features
struct AudioFeatures {
    int* segmentDuration = nullptr; // Duration per segment (seconds)
    double* SPLrms = nullptr; // SPLrms
    double* SPLpk = nullptr; // Peak SPL
    double* impulsivity = nullptr; // Kurtosis
    double* dissim = nullptr; // Dissimilarity between segments
    int* peakCount = nullptr; // # of peaks
    double** autocorr = nullptr; // Autocorrelation matrix

    // # of segments
    int segmentDurationLen = 0;
    int SPLrmsLen = 0;
    int SPLpkLen = 0;
    int impulsivityLen = 0;
    int dissimLen = 0;
    int peakCountLen = 0;
    int autocorrRows = 0; // Time segments
    int autocorrCols = 0; // Lags
};

struct AudioData {
    double** samples; // 2D array of audio samples [channel][frame]
    int numChannels; // # of audio channels
    int numFrames; // # of frames per channel
    int sampleRate; // Sampling rate (Hz)
    double duration; // Duration of audio (seconds)
};

// Periodicity / impulsivity
struct SoloPer {
    int* peakCount; // # of peaks per time window
    double** autocorr; // Autocorrelation per segment
    int peakcountLength; // Length of peakcount array
    int autocorrRows; // # of time windows - rows
    int autocorrCols; // # of lags - columns
};

struct ArrayShiftFFT {
    double* data; // Array of samples after shift
    int length; // Length of array

    // Destructor
    ~ArrayShiftFFT() { delete[] data; }
};

// FFTW complex buffer + plan
struct FFTWHandler {
    fftw_complex* buf = nullptr; // Buffer for FFT computation: Time to frequency
    fftw_plan forwardPlan = nullptr; // Forward FFT plan: Frequency to time
    fftw_plan inversePlan = nullptr; // Inverse FFT plan
    int size = 0; // # of points in FFT

    // Constructor
    FFTWHandler(int N) : size(N) {
        buf = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * N);
        if (!buf) { throw bad_alloc(); }

        forwardPlan = fftw_plan_dft_1d(size, buf, buf, FFTW_FORWARD, FFTW_ESTIMATE);
        if (!forwardPlan) { // Plan failed to create
            fftw_free(buf); // Free memory
            throw runtime_error("FFTW forward plan creation failed");
        }

        inversePlan = fftw_plan_dft_1d(size, buf, buf, FFTW_BACKWARD, FFTW_ESTIMATE);
        if (!inversePlan) { // Plan failed to create
            fftw_destroy_plan(forwardPlan); // Free memory
            fftw_free(buf);
            throw runtime_error("FFTW inverse plan creation failed");
        }
    }

    // Destructor
    ~FFTWHandler() {
        if (forwardPlan) { fftw_destroy_plan(forwardPlan); }
        if (inversePlan) { fftw_destroy_plan(inversePlan); }
        if (buf) { fftw_free(buf); }
    }
};

// Hold extracted time information per file
struct FileTimeInfo {
    tm baseTime; // Struct containing date/time components
    bool timeExtracted; // Time successfully extracted
    string filename;
};

struct ThreadArgs { // Worker threads for parallel processing
    atomic<int>* nextIndex; // Counter for thread-safe file indexing
    int totalFiles; // # of audio files to process
    char (*filePaths)[128]; // Input file paths
    AudioFeatures* allFeatures; // Feature extraction results
    char (*filenames)[128]; // Names of files
    FileTimeInfo* fileTimeInfo;

    // User-given arguments
    int numBits, peakVolts, timeWin, fftWin, arti, fLow, fHigh, downSample;
    double RS, avTime;
    bool omitPartialMinute;
};

// Replaces backslashes with forward slashes to work with Windows file paths
string fixFilePath(const string& path) {
    string fixedPath = path;
    replace(fixedPath.begin(), fixedPath.end(), '\\', '/'); // Replace '\\' with '/'
    return fixedPath;
}

// Read audio samples from files
AudioData audioRead(const string& filename, SampleRange range = { 1, -1 }) {
    SF_INFO sfinfo = {}; // Audio metadata (# of channels, sample rate, etc.)
    SNDFILE* file = sf_open(filename.c_str(), SFM_READ, &sfinfo); // Open file for reading

    if (!file) { throw runtime_error("Error opening audio file: " + string(sf_strerror(file))); } // File open unsuccessful

    // Sample range calculation to fit within file bounds
    int totalFrames = sfinfo.frames;
    int endSample;
    if (range.endSample == -1) { endSample = totalFrames; }
    else { endSample = min(range.endSample, totalFrames); }
    int startSample = max(0, range.startSample - 1);
    int numFramesToRead = endSample - startSample;

    if (numFramesToRead <= 0) { // Range invalid
        sf_close(file);
        throw runtime_error("Invalid sample range");
    }

    sf_seek(file, startSample, SEEK_SET); // Adjust file position to startSample

    int numChannels = sfinfo.channels;
    double* interleavedSamples = new double[numFramesToRead * numChannels]; // Raw interleaved samples

    int format = sfinfo.format & SF_FORMAT_SUBMASK; // Extract audio subtype from full format 

    // Read / convert audio samples
    switch (format) { // Convert samples according to bit format
    case SF_FORMAT_PCM_16: {
        short* tempBuffer = new short[numFramesToRead * numChannels];
        sf_readf_short(file, tempBuffer, numFramesToRead);
        for (int i = 0; i < numFramesToRead * numChannels; ++i) {
            interleavedSamples[i] = static_cast<double>(tempBuffer[i]);
        }
        delete[] tempBuffer;
        break;
    }
    case SF_FORMAT_PCM_24:
    case SF_FORMAT_PCM_32: {
        int* tempBuffer = new int[numFramesToRead * numChannels];
        sf_readf_int(file, tempBuffer, numFramesToRead);
        for (int i = 0; i < numFramesToRead * numChannels; ++i) {
            interleavedSamples[i] = static_cast<double>(tempBuffer[i]);
        }
        delete[] tempBuffer;
        break;
    }
    default: // Bit format invalid
        sf_close(file);
        delete[] interleavedSamples;
        throw runtime_error("Unsupported bit format");
    }

    sf_close(file); // Close file

    // Deinterleave samples into separate channels
    double** samples = new double* [numChannels];
    for (int ch = 0; ch < numChannels; ++ch) { // Allocate memory per channel
        samples[ch] = new double[numFramesToRead];
    }

    for (int i = 0; i < numFramesToRead; ++i) { // Populate records for each channel
        for (int ch = 0; ch < numChannels; ++ch) {
            samples[ch][i] = interleavedSamples[i * numChannels + ch];
        }
    }

    delete[] interleavedSamples; // Deallocate memory

    double duration = static_cast<double>(sfinfo.frames) / sfinfo.samplerate; // Recording length (seconds)

    return AudioData{ samples, numChannels, numFramesToRead, sfinfo.samplerate, duration };
}

__global__ void downsampleKernel(const double* x, double* result, int length, int factor) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index of thread
    // Determine if downsampled data is in input bounds
    if (index * factor < length) { result[index] = x[index * factor]; } // Compute output sample if within bounds
}

// Reduce sampling rate to lower frequency
double* downSample(const double* x, int length, int factor, int& newLength) {
    if (factor <= 0) { throw invalid_argument("Factor must be positive"); } // Validate input

    newLength = (length + factor - 1) / factor; // # of samples in downsampled signal

    // Allocate GPU memory
    double* deviceInput;
    double* deviceOutput;
    hipMalloc(&deviceInput, sizeof(double) * length);
    hipMalloc(&deviceOutput, sizeof(double) * newLength);

    // Copy input signal from host to device
    hipMemcpy(deviceInput, x, sizeof(double) * length, hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256; // # of threads per block
    int blocks = (newLength + threads - 1) / threads; // # of blocks
    downsampleKernel <<<blocks, threads >>> (deviceInput, deviceOutput, length, factor);

    double* result = new double[newLength]; // Allocate memory on host
    // Copy downsampled result to host
    hipMemcpy(result, deviceOutput, sizeof(double) * newLength, hipMemcpyDeviceToHost);

    // Deallocate GPU memory
    hipFree(deviceInput);
    hipFree(deviceOutput);

    return result; // Downsampled signal
}

__global__ void fftShiftKernel(const double* input, double* shifted, int length) {
    // Shift array to center around zero index component
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global index
    if (i < length) { shifted[i] = input[(i + (length / 2)) % length]; } // Process valid indices
}

// CUDA kernel to compute shifted frequency array / apply bandpass filter
__global__ void applyBandpassFilter(hipfftDoubleComplex* freqData, int numPoints,
            double freqStep, double fLow, double fHigh) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Thread indexing and bounds checking
    if (i >= numPoints) { return; }

    int shiftedIndex = (i + numPoints / 2) % numPoints; // Calculate index

    double freq = (shiftedIndex - numPoints / 2) * freqStep; // Convert index to frequency with index centering
    double absFreq = fabs(freq);

    // Zero out components outside specified range
    if (absFreq < fLow || absFreq > fHigh) {
        freqData[i].x = 0.0;
        freqData[i].y = 0.0;
    }
}

// CUDA kernel to normalize inverse FFT / compute amplitude spectrum
__global__ void normalizeAndComputeAmplitude(const hipfftDoubleComplex* timeData, double* outputTime,
            double* outputAmp, int numPoints) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute thread global index
    if (i >= numPoints) { return; } // Ensure valid thread range

    double norm = 1.0 / numPoints;
    outputTime[i] = timeData[i].x * norm; // Normalize inverse FFT real output
    // Compute magnitude at each point
    outputAmp[i] = sqrt(timeData[i].x * timeData[i].x + timeData[i].y * timeData[i].y);
}

// Apply bandpass filter in frequency domain
BandpassFilter bandpassFilter(const double* timeSeries, int numPts, double frequency, double flow, double freqHigh) {
    // Allocate memory on device
    hipfftDoubleComplex* deviceFreqData;
    hipMalloc(&deviceFreqData, sizeof(hipfftDoubleComplex) * numPts);

    // Copy real timeSeries to device
    hipfftDoubleComplex* hostInput = new hipfftDoubleComplex[numPts]; // Convert to complex
    for (int i = 0; i < numPts; ++i) {
        hostInput[i].x = timeSeries[i];
        hostInput[i].y = 0.0;
    }

    // Copy frequency data to device
    hipMemcpy(deviceFreqData, hostInput, sizeof(hipfftDoubleComplex) * numPts, hipMemcpyHostToDevice);
    delete[] hostInput; // Free host memory

    // Execute forward FFT
    hipfftHandle planForward;
    hipfftPlan1d(&planForward, numPts, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(planForward, deviceFreqData, deviceFreqData, HIPFFT_FORWARD);

    // Apply bandpass filter
    double recordingLen = numPts * frequency; // Time span (seconds)
    double freqStep = 1.0 / recordingLen;
    if (freqHigh == 0.0) { freqHigh = 0.5 / frequency; }

    // Zero out frequencies outside given range
    int threads = 256;
    int blocks = (numPts + threads - 1) / threads;
    applyBandpassFilter <<<blocks, threads >>> (deviceFreqData, numPts, freqStep, flow, freqHigh);
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution

    // Execute inverse FFT
    hipfftExecZ2Z(planForward, deviceFreqData, deviceFreqData, HIPFFT_BACKWARD); // Reuse plan

    // Allocate output arrays
    double* deviceTimeOut;
    double* deviceAmplitudeOut;
    hipMalloc(&deviceTimeOut, sizeof(double) * numPts);
    hipMalloc(&deviceAmplitudeOut, sizeof(double) * numPts);

    normalizeAndComputeAmplitude <<<blocks, threads >>> (deviceFreqData, deviceTimeOut, deviceAmplitudeOut, numPts);
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution

    // Copy results to host
    double* timeSeriesFilt = new double[numPts];
    double* amplitudeSpectrum = new double[numPts];
    // Copy magnitude spectrum / time series data to host
    hipMemcpy(timeSeriesFilt, deviceTimeOut, sizeof(double) * numPts, hipMemcpyDeviceToHost);
    hipMemcpy(amplitudeSpectrum, deviceAmplitudeOut, sizeof(double) * numPts, hipMemcpyDeviceToHost);

    // Cleanup
    hipfftDestroy(planForward);
    hipFree(deviceFreqData);
    hipFree(deviceTimeOut);
    hipFree(deviceAmplitudeOut);

    return BandpassFilter(timeSeriesFilt, amplitudeSpectrum, numPts);
}

__global__ void partialSumsKernel(const double* data, double* sumOut, double* sumSqOut, int pointsPerTimeWin) {
    // Shared memory per thread block
    __shared__ double localSum[256];
    __shared__ double localSumSq[256];

    // Thread global index
    int threadIndex = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    double val; // Load current val
    if (index < pointsPerTimeWin) { val = data[index]; }
    else { val = 0.0; }
    // Write data / squared value into shared memory
    localSum[threadIndex] = val;
    localSumSq[threadIndex] = val * val;
    __syncthreads(); // Ensure all threads have written

    // Reduce within block
    for (int stepSize = blockDim.x / 2; stepSize > 0; stepSize >>= 1) { // Halves active threads on each pass
        if (threadIndex < stepSize) {
            // Calculate sums
            localSum[threadIndex] += localSum[threadIndex + stepSize];
            localSumSq[threadIndex] += localSumSq[threadIndex + stepSize];
        }
        __syncthreads(); // Ensure all threads completed before next step
    }

    // First thread per block writes result to global memory
    if (threadIndex == 0) {
        sumOut[blockIdx.x] = localSum[0]; // Block sum
        sumSqOut[blockIdx.x] = localSumSq[0]; // Block sum of squares
    }
}

__global__ void fourthMomentKernel(const double* data, double* fourthOut, double mean, int pointsPerTimeWin) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global index
    if (i >= pointsPerTimeWin) { return; } // Protect against out of bounds threads

    // Fourth central moment
    double centered = data[i] - mean;
    fourthOut[i] = centered * centered * centered * centered;
}

__global__ void reduceSumKernel(const double* input, double* output, int n) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load shared memory with either data or 0
    if (i < n) { sdata[tid] = input[i]; }
    else { sdata[tid] = 0.0; }
    __syncthreads();

    // Reduce in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) { sdata[tid] += sdata[tid + s]; }
        __syncthreads();
    }

    // Write block result to global memory
    if (tid == 0) {  output[blockIdx.x] = sdata[0]; }
}

// Calculate kurtosis used for impulsivity of a signal
double calculateKurtosis(const double* hostData, int pointsPerTimeWin) {
    if (pointsPerTimeWin <= 0 || hostData == nullptr) { throw invalid_argument("Input array is empty or null"); }

    // Allocate device memory
    double* deviceData, * deviceSumPartial, * deviceSumSqPartial, * deviceFourth;
    hipMalloc(&deviceData, pointsPerTimeWin * sizeof(double));
    hipMemcpy(deviceData, hostData, pointsPerTimeWin * sizeof(double), hipMemcpyHostToDevice); // Copy data to device

    // Thread / block setup
    int threads = 256;
    int blocks = (pointsPerTimeWin + threads - 1) / threads;

    hipMalloc(&deviceSumPartial, blocks * sizeof(double)); // Allocate device memory for partial sum
    hipMalloc(&deviceSumSqPartial, blocks * sizeof(double)); // Allocate device memory for partial sum square

    // Compute mean / variance components
    partialSumsKernel <<<blocks, threads >>> (deviceData, deviceSumPartial, deviceSumSqPartial, pointsPerTimeWin);
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution

    // Reduce partial results to final results
    double* sumReduceBuf = deviceSumPartial;
    int sumN = blocks;
    while (sumN > 1) {
        int sumBlocks = (sumN + threads - 1) / threads;
        reduceSumKernel << <sumBlocks, threads, threads * sizeof(double) >> > (sumReduceBuf, sumReduceBuf, sumN);
        hipDeviceSynchronize();
        sumN = sumBlocks;
    }

    // Sum values in parallel
    double totalSum;
    hipMemcpy(&totalSum, sumReduceBuf, sizeof(double), hipMemcpyDeviceToHost);

    double* sumSqReduceBuf = deviceSumSqPartial;
    int sumSqN = blocks;
    while (sumSqN > 1) {
        int sumBlocks = (sumSqN + threads - 1) / threads;
        reduceSumKernel << <sumBlocks, threads, threads * sizeof(double) >> > (sumSqReduceBuf, sumSqReduceBuf, sumSqN);
        hipDeviceSynchronize();
        sumSqN = sumBlocks;
    }

    double totalSumSq;
    hipMemcpy(&totalSumSq, sumSqReduceBuf, sizeof(double), hipMemcpyDeviceToHost);

    double mean = totalSum / pointsPerTimeWin; // Calculate mean
    double variance = (totalSumSq / pointsPerTimeWin) - (mean * mean); // Calculate variance

    if (variance < 1e-12) { // Avoid divide by zero
        hipFree(deviceData);
        hipFree(deviceSumPartial);
        hipFree(deviceSumSqPartial);
        return 0.0;
    }

    // Compute fourth moment
    hipMalloc(&deviceFourth, pointsPerTimeWin * sizeof(double));
    fourthMomentKernel <<<blocks, threads >>> (deviceData, deviceFourth, mean, pointsPerTimeWin);
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution

    // Reduce fourth moment array to scalar value / sum values in parallel
    double* fourthReduceBuf = deviceFourth;
    int fourthN = pointsPerTimeWin;
    while (fourthN > 1) {
        int sumBlocks = (fourthN + threads - 1) / threads;
        reduceSumKernel << <sumBlocks, threads, threads * sizeof(double) >> > (fourthReduceBuf, fourthReduceBuf, fourthN);
        hipDeviceSynchronize();
        fourthN = sumBlocks;
    }

    double fourthMoment;
    hipMemcpy(&fourthMoment, fourthReduceBuf, sizeof(double), hipMemcpyDeviceToHost);
    fourthMoment /= pointsPerTimeWin;

    // Cleanup
    hipFree(deviceData);
    hipFree(deviceSumPartial);
    hipFree(deviceSumSqPartial);
    hipFree(deviceFourth);

    return fourthMoment / (variance * variance); // Kurtosis value
}

// CUDA kernel for envelope calculation
__global__ void envelopeKernel(const hipfftDoubleComplex* hilbert, double* envelope, int pointsPerTimeWin) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute magnitude for idx
    if (index < pointsPerTimeWin)
        { envelope[index] = sqrt(hilbert[index].x * hilbert[index].x + hilbert[index].y * hilbert[index].y); }
}

// CUDA kernel for correlation computation
__global__ void correlationKernel(const double* real, const double* imaginary, int seriesLength,
            double* corrVals, int maxLag, int offset) {
    int lag = blockIdx.x * blockDim.x + threadIdx.x;
    if (lag > maxLag) { return; } // Skip out of range lags
    
    // Accumulators for statistical calculations
    double sumReal = 0.0, sumImagninary = 0.0, sumRealSquare = 0.0, sumImaginarySquare = 0.0, sumRealImaginaryProd = 0.0;
    int sampleCount = 0;
    
    // Loop through overlapping samples of current lag
    for (int i = 0; i < seriesLength - (lag + offset); i++) {
        double realVal = real[i];
        double imaginaryVal = imaginary[i + lag + offset];
        
        if (!isnan(realVal) && !isnan(imaginaryVal)) { // Skip NaNs
            sumReal += realVal;
            sumImagninary += imaginaryVal;
            sumRealSquare += realVal * realVal;
            sumImaginarySquare += imaginaryVal * imaginaryVal;
            sumRealImaginaryProd += realVal * imaginaryVal;
            sampleCount++;
        }
    }
    
    if (sampleCount == 0) { // No valid samples
        corrVals[lag] = NAN;
        return;
    }
    
    // Means / variances
    double meanReal = sumReal / sampleCount;
    double meanImaginary = sumImagninary / sampleCount;
    double meanXSquare = sumRealSquare / sampleCount;
    double meanYSquare = sumImaginarySquare / sampleCount;
    
    double covar = (sumRealImaginaryProd / sampleCount) - (meanReal * meanImaginary); // Covariance
    double denomReal = sqrt(meanXSquare - (meanReal * meanReal));
    double denomImaginary = sqrt(meanYSquare - (meanImaginary * meanImaginary)); // Calculate standard deviation
    
    // Normalize correlation
    if (denomReal == 0.0 || denomImaginary == 0.0) { corrVals[lag] = NAN; }
    else { corrVals[lag] = covar / (denomReal * denomImaginary); }
}

// CUDA kernel for FFT magnitude calculation
__global__ void fftMagnitudeKernel(const hipfftDoubleComplex* fftData, double* magnitude, int pointsPerFFT) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate magnitude
    if (idx < pointsPerFFT) { magnitude[idx] = sqrt(fftData[idx].x * fftData[idx].x + fftData[idx].y * fftData[idx].y); }
}

// GPU-accelerated correlation function
Correlation correl5GPU(const double* timeSeries1, const double* timeSeries2, 
            int seriesLength, int lags, int offset) {
    int len = lags + 1; // # of correlation values to calculate
    
    // Allocate GPU memory
    double * deviceInputSignal1, * deviceInputSignal2, * deviceCorrVals;
    hipMalloc(&deviceInputSignal1, sizeof(double) * seriesLength);
    hipMalloc(&deviceInputSignal2, sizeof(double) * seriesLength);
    hipMalloc(&deviceCorrVals, sizeof(double) * len);
    
    // Copy deviceInputSignal1 / deviceInputSignal2 to device
    hipMemcpy(deviceInputSignal1, timeSeries1, sizeof(double) * seriesLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceInputSignal2, timeSeries2, sizeof(double) * seriesLength, hipMemcpyHostToDevice);
    
    // Launch correlation kernel
    // Specify dimensions
    dim3 block(256);
    dim3 grid((len + block.x - 1) / block.x); // All data points covered by threads
    correlationKernel<<<grid, block>>>(deviceInputSignal1, deviceInputSignal2, seriesLength, deviceCorrVals, lags, offset);
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution
    
    // Allocate host memory
    double* corrVals = new double[len];
    double* lagVals = new double[len];
    
    hipMemcpy(corrVals, deviceCorrVals, sizeof(double) * len, hipMemcpyDeviceToHost); // Copy corrVals to host
    
    // Fill lag values
    for (int i = 0; i < len; ++i) { lagVals[i] = static_cast<double>(i); }
    
    // Cleanup GPU memory
    hipFree(deviceInputSignal1);
    hipFree(deviceInputSignal2);
    hipFree(deviceCorrVals);
    
    return Correlation(corrVals, lagVals, len);
}

// Kernel to square / segment the input
__global__ void squareAndSegment(const double* input, double* output, int sampWindowSize, int numTimeWins) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    int total = sampWindowSize * numTimeWins; // # of samples

    if (index < total) { // Ensure index in bounds
        double val = input[index];
        output[index] = val * val; // Signal energy
    }
}

// Kernel to average squared values
__global__ void computeAverages(const double* squared, double* outputAvg, int sampWindowSize,
            int avgWinSize, int numavwin, int numTimeWins) {
    int segIndex = blockIdx.x; // Segment index
    int win = threadIdx.x; // Averaging window index

    if (segIndex < numTimeWins && win < numavwin) { // Check bounds
        int baseIndex = segIndex * sampWindowSize + win * avgWinSize;
        double sum = 0.0;
        for (int i = 0; i < avgWinSize; i++) { sum += squared[baseIndex + i]; } // Sum of samples in window
        outputAvg[segIndex * numavwin + win] = sum / avgWinSize; // Mean in window
    }
}

// Calculate autocorrelation / peak counts
SoloPer calculatePeriodicity(const double* pFiltInput, int inputLength, double fs, double timewin, double avtime) {
    // Calculate window sizes
    int sampWindowSize = static_cast<int>(fs * timewin); // # of samples in time window
    int numTimeWins = inputLength / sampWindowSize; // # of time windows
    if (numTimeWins == 0) { throw runtime_error("Empty time window"); }

    int totalSamples = sampWindowSize * numTimeWins;
    int avgWinSize = static_cast<int>(fs * avtime); // Samples in averaging window
    int numAvWin = sampWindowSize / avgWinSize; // Averaging windows per time window

    // GPU memory allocations
    double* deviceInput, * deviceSquared, * deviceAvg;
    hipMalloc(&deviceInput, totalSamples * sizeof(double));
    hipMalloc(&deviceSquared, totalSamples * sizeof(double));
    hipMalloc(&deviceAvg, numTimeWins * numAvWin * sizeof(double));

    hipMemcpy(deviceInput, pFiltInput, totalSamples * sizeof(double), hipMemcpyHostToDevice); // Copy input to device

    // Launch square / segment kernel
    int threads = 256;
    int blocks = (totalSamples + threads - 1) / threads;
    squareAndSegment <<<blocks, threads >>> (deviceInput, deviceSquared, sampWindowSize, numTimeWins);

    // Calculate average of squared values
    computeAverages <<<numTimeWins, numAvWin >>> (deviceSquared, deviceAvg, sampWindowSize,
            avgWinSize, numAvWin, numTimeWins);

    // Copy averages back to host
    double* hostAvg = new double[numTimeWins * numAvWin];
    hipMemcpy(hostAvg, deviceAvg, numTimeWins * numAvWin * sizeof(double), hipMemcpyDeviceToHost);

    // Outputs for correlation / peak count
    int pAvTotRows = numAvWin;
    int lagLimit = static_cast<int>(pAvTotRows * 0.7); // 70% of lags
    int pAvTotCols = numTimeWins;

    double** acorr = new double* [pAvTotCols]; // Autocorrelation per window
    int* pkcount = new int[pAvTotCols]; // Peak count per window

    // Iterate through time windows - Calculate autocorr / peak count
    for (int i = 0; i < pAvTotCols; i++) {
        // Calculate correlation
        hipSetDevice(0);
        Correlation corrResult = correl5GPU(&hostAvg[i * numAvWin], &hostAvg[i * numAvWin], pAvTotRows, lagLimit, 0);
        acorr[i] = new double[lagLimit + 1];
        for (int j = 0; j <= lagLimit; ++j) { acorr[i][j] = corrResult.correlationValues[j]; }

        // Calculate peak count
        int peakCount = 0;
        for (int j = 1; j < lagLimit; j++) {
            if (acorr[i][j] > acorr[i][j - 1] && acorr[i][j] > acorr[i][j + 1]) {
                // Find min to left / right to find prominence
                double leftMin = acorr[i][j];
                for (int k = j - 1; k >= 0 && acorr[i][k] < acorr[i][j]; k--) { leftMin = min(leftMin, acorr[i][k]); }
                double rightMin = acorr[i][j];
                for (int k = j + 1; k <= lagLimit && acorr[i][k] < acorr[i][j]; k++)
                    { rightMin = min(rightMin, acorr[i][k]); }
                double prominence = acorr[i][j] - max(leftMin, rightMin);
                if (prominence > 0.5) { peakCount++; } // Threshold reached
            }
        }
        pkcount[i] = peakCount;
    }

    // Free GPU memory
    hipFree(deviceInput);
    hipFree(deviceSquared);
    hipFree(deviceAvg);
    delete[] hostAvg;

    // Return result
    SoloPer result;
    result.peakCount = pkcount;
    result.autocorr = acorr;
    result.peakcountLength = pAvTotCols;
    result.autocorrRows = pAvTotCols;
    result.autocorrCols = lagLimit + 1;

    return result;
}

// Zeroes out negative frequencies / doubles positive frequencies to create frequency signal
__global__ void hilbertFilterKernel(hipfftDoubleComplex* data, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len) { return; } // Ensure index in valid range
    
    int half = len / 2; // Half way point of FFT
    int upper;
    if (len % 2 == 0) { upper = half - 1; }
    else { upper = half; }
    
    if (index >= 1 && index <= upper) {
        // Multiply positive frequencies by 2
        data[index].x *= 2.0;
        data[index].y *= 2.0;
    } else if (index > half) {
        // Zero out negative frequencies
        data[index].x = 0.0;
        data[index].y = 0.0;
    }
}

// Convert real input to complex array
__global__ void initializeComplex(double* input, hipfftDoubleComplex* output, int len) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    if (threadIndex < len) {
        output[threadIndex].x = input[threadIndex]; // Real
        output[threadIndex].y = 0.0; // Imaginary
    }
}

// Normalize inverse FFT result to preserve signal amplitude
__global__ void normalizeResult(hipfftDoubleComplex* data, int len) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    if (threadIndex < len) {
        data[threadIndex].x /= len; // Normalize real part
        data[threadIndex].y /= len; // Normalize imaginary part
    }
}

// Converts real values to complex analytic signal using FFT
fftw_complex* hilbertRawGPU(const double* input, int inputLen) {
    if (!input || inputLen <= 0) { // Validate input
        cerr << "Invalid input\n";
        return nullptr;
    }

    hipfftDoubleComplex* deviceData = nullptr; // Final complex array
    hipMalloc(&deviceData, sizeof(hipfftDoubleComplex) * inputLen);

    double* deviceIinput = nullptr; // Device buffer for input
    hipMalloc(&deviceIinput, sizeof(double) * inputLen);
    hipMemcpy(deviceIinput, input, sizeof(double) * inputLen, hipMemcpyHostToDevice); // Copy input to device

    // Convert to complex
    dim3 block(256);
    dim3 grid((inputLen + block.x - 1) / block.x); // All data points covered by threads
    initializeComplex <<<grid, block >>> (deviceIinput, deviceData, inputLen);
    hipFree(deviceIinput); // Free deviceIinput early

    hipfftHandle plan; // FFT plan
    hipfftPlan1d(&plan, inputLen, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan, deviceData, deviceData, HIPFFT_FORWARD); // Apply FFT: Time to frequency

    hilbertFilterKernel <<<grid, block >>> (deviceData, inputLen); // Apply filter
    hipDeviceSynchronize(); // Ensure previous operations are completed before execution

    hipfftExecZ2Z(plan, deviceData, deviceData, HIPFFT_BACKWARD); // Apply inverse FFT: Frequency to time

    // Normalize to preserve amplitude
    normalizeResult <<<grid, block >>> (deviceData, inputLen);

    // Copy result to host
    fftw_complex* result = (fftw_complex*)fftw_malloc(sizeof(fftw_complex) * inputLen);
    if (!result) { // Unsuccessful memory allocation
        cerr << "Host allocation failed\n";
        hipFree(deviceData);
        hipfftDestroy(plan);
        return nullptr;
    }

    // Copy results to host
    hipMemcpy(result, deviceData, sizeof(hipfftDoubleComplex) * inputLen, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(deviceData);
    hipfftDestroy(plan);

    return result;
}

// Perform per-window envelope comparisons with hilbert transofrms / FFTs to calculate dissimilarity
double* calculateDissimGPU(double** timechunkMatrix, int ptsPerTimewin, int numTimeWin,
            double fftWin, double fs, int& outLen) {

    // # of FFT points
    int ptsPerFFT = static_cast<int>(fftWin * fs);
    if (ptsPerFFT <= 0 || ptsPerTimewin <= 0 || numTimeWin <= 1) {
        outLen = 0;
        return nullptr;
    }

    // # of overlapping FFT windows per time window
    int numfftwin = (ptsPerTimewin - ptsPerFFT) / ptsPerFFT + 1;
    if (numfftwin <= 0) {
        outLen = 0;
        return nullptr;
    }

    outLen = numTimeWin; // # of outputs

    // CUFFT plan
    hipfftHandle fftPlan;
    if (hipfftPlan1d(&fftPlan, ptsPerFFT, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS) {
        outLen = 0;
        return nullptr;
    }

    // Allocate GPU buffers
    hipfftDoubleComplex* deviceFFTInput = nullptr;
    hipfftDoubleComplex* deviceFFTOutput = nullptr;
    double* deviceMagnitude = nullptr;
    double* deviceEnvelope1 = nullptr;
    double* deviceEnvelope2 = nullptr;

    // Free partial allocations on failure
    if (hipMalloc(&deviceFFTInput, sizeof(hipfftDoubleComplex) * ptsPerFFT) != hipSuccess ||
        hipMalloc(&deviceFFTOutput, sizeof(hipfftDoubleComplex) * ptsPerFFT) != hipSuccess ||
        hipMalloc(&deviceMagnitude, sizeof(double) * ptsPerFFT) != hipSuccess ||
        hipMalloc(&deviceEnvelope1, sizeof(double) * ptsPerTimewin) != hipSuccess ||
        hipMalloc(&deviceEnvelope2, sizeof(double) * ptsPerTimewin) != hipSuccess) {

        hipFree(deviceFFTInput);
        hipFree(deviceFFTOutput);
        hipFree(deviceMagnitude);
        hipFree(deviceEnvelope1);
        hipFree(deviceEnvelope2);
        hipfftDestroy(fftPlan);
        outLen = 0;
        return nullptr;
    }

    // Allocate host result array
    double* diss = new double[outLen];

    diss[0] = NAN; // No previous record for comparison

    // Temporary buffers for hilbert / fft inputs
    hipfftDoubleComplex* hil1Host = new hipfftDoubleComplex[ptsPerTimewin];
    hipfftDoubleComplex* hil2Host = new hipfftDoubleComplex[ptsPerTimewin];
    hipfftDoubleComplex* fftInputHost = new hipfftDoubleComplex[ptsPerFFT];
    double* envelope1Host = new double[ptsPerTimewin];
    double* envelope2Host = new double[ptsPerTimewin];
    double* magnitudeHost = new double[ptsPerFFT];
    double* fftAHost = new double[ptsPerFFT];
    double* fftBHost = new double[ptsPerFFT];

    // Specify dimensions
    dim3 block(256);
    dim3 gridFFT((ptsPerFFT + block.x - 1) / block.x); // All data points covered by threads
    dim3 gridEnv((ptsPerTimewin + block.x - 1) / block.x);

    // Iterate over adjacent pairs of time chunks
    for (int i = 1; i < outLen; ++i) { // diss[0] already NAN
        // Calculate analytic signals
        hipSetDevice(0); // Use first available GPU
        fftw_complex* hil1 = hilbertRawGPU(timechunkMatrix[i - 1], ptsPerTimewin);
        hipSetDevice(0); // Use first available GPU
        fftw_complex* hil2 = hilbertRawGPU(timechunkMatrix[i], ptsPerTimewin);

        if (!hil1 || !hil2) { // Failed hilbert computations
            diss[i] = NAN;
            if (hil1) fftw_free(hil1);
            if (hil2) fftw_free(hil2);
            continue;
        }

        // Copy hilbert result to host hipfftDoubleComplex arrays
        for (int k = 0; k < ptsPerTimewin; ++k) {
            hil1Host[k].x = hil1[k][0];
            hil1Host[k].y = hil1[k][1];
            hil2Host[k].x = hil2[k][0];
            hil2Host[k].y = hil2[k][1];
        }

        fftw_free(hil1);
        fftw_free(hil2);

        // Copy hilbert data to device
        hipfftDoubleComplex* deviceHil1 = nullptr;
        hipfftDoubleComplex* deviceHil2 = nullptr;
        hipMalloc(&deviceHil1, sizeof(hipfftDoubleComplex) * ptsPerTimewin);
        hipMalloc(&deviceHil2, sizeof(hipfftDoubleComplex) * ptsPerTimewin);
        hipMemcpy(deviceHil1, hil1Host, sizeof(hipfftDoubleComplex) * ptsPerTimewin, hipMemcpyHostToDevice);
        hipMemcpy(deviceHil2, hil2Host, sizeof(hipfftDoubleComplex) * ptsPerTimewin, hipMemcpyHostToDevice);

        // Compute amplitude envelopes on device
        envelopeKernel <<<gridEnv, block >>> (deviceHil1, deviceEnvelope1, ptsPerTimewin);
        envelopeKernel <<<gridEnv, block >>> (deviceHil2, deviceEnvelope2, ptsPerTimewin);
        hipDeviceSynchronize(); // Ensure previous operations are completed before execution

        // Copy envelopes to host
        hipMemcpy(envelope1Host, deviceEnvelope1, sizeof(double) * ptsPerTimewin, hipMemcpyDeviceToHost);
        hipMemcpy(envelope2Host, deviceEnvelope2, sizeof(double) * ptsPerTimewin, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(deviceHil1);
        hipFree(deviceHil2);

        // Normalize envelopes
        double sum1 = 0.0;
        double sum2 = 0.0;
        for (int j = 0; j < ptsPerTimewin; ++j) {
            sum1 += envelope1Host[j];
            sum2 += envelope2Host[j];
        }
        // Amplitude between records
        if (sum1 > 1e-12) {
            for (int j = 0; j < ptsPerTimewin; ++j) { envelope1Host[j] /= sum1; }
        }
        if (sum2 > 1e-12) {
            for (int j = 0; j < ptsPerTimewin; ++j) { envelope2Host[j] /= sum2; }
        }

        // Calculate time dissimilarity
        double timeDiss = 0.0;
        for (int j = 0; j < ptsPerTimewin; ++j) { timeDiss += fabs(envelope1Host[j] - envelope2Host[j]); }
        timeDiss *= 0.5;

        // Initialize fftAHost / fftBHost arrays to zero
        for (int j = 0; j < ptsPerFFT; ++j) {
            fftAHost[j] = 0.0;
            fftBHost[j] = 0.0;
        }

        // Frequency domain dissimilarity
        for (int w = 0; w < numfftwin; ++w) {
            int base = w * ptsPerFFT;

            // Process first time chunk FFT
            for (int j = 0; j < ptsPerFFT; ++j) {
                fftInputHost[j].x = timechunkMatrix[i][base + j];
                fftInputHost[j].y = 0.0;
            }
            hipMemcpy(deviceFFTInput, fftInputHost, sizeof(hipfftDoubleComplex) * ptsPerFFT, hipMemcpyHostToDevice);
            hipfftExecZ2Z(fftPlan, deviceFFTInput, deviceFFTOutput, HIPFFT_FORWARD);

            fftMagnitudeKernel <<<gridFFT, block >>> (deviceFFTOutput, deviceMagnitude, ptsPerFFT);
            hipDeviceSynchronize(); // Ensure previous operations are completed before execution

            hipMemcpy(magnitudeHost, deviceMagnitude, sizeof(double) * ptsPerFFT, hipMemcpyDeviceToHost);

            for (int j = 0; j < ptsPerFFT; ++j) { fftAHost[j] += magnitudeHost[j]; }

            // Process second time chunk FFT
            for (int j = 0; j < ptsPerFFT; ++j) {
                fftInputHost[j].x = timechunkMatrix[i - 1][base + j];
                fftInputHost[j].y = 0.0;
            }
            hipMemcpy(deviceFFTInput, fftInputHost, sizeof(hipfftDoubleComplex) * ptsPerFFT, hipMemcpyHostToDevice);
            hipfftExecZ2Z(fftPlan, deviceFFTInput, deviceFFTOutput, HIPFFT_FORWARD);

            fftMagnitudeKernel <<<gridFFT, block >>> (deviceFFTOutput, deviceMagnitude, ptsPerFFT);
            hipDeviceSynchronize(); // Ensure previous operations are completed before execution

            hipMemcpy(magnitudeHost, deviceMagnitude, sizeof(double) * ptsPerFFT, hipMemcpyDeviceToHost);

            for (int j = 0; j < ptsPerFFT; ++j) { fftBHost[j] += magnitudeHost[j]; }
        }

        // Normalize frequency spectra
        double totalA = 0.0;
        double totalB = 0.0;
        for (int j = 0; j < ptsPerFFT; ++j) {
            totalA += fftAHost[j];
            totalB += fftBHost[j];
        }

        // Populate fft hosts
        if (totalA > 1e-12) {
            for (int j = 0; j < ptsPerFFT; ++j) { fftAHost[j] /= totalA; }
        }
        if (totalB > 1e-12) {
            for (int j = 0; j < ptsPerFFT; ++j) { fftBHost[j] /= totalB; }
        }

        // Calculate frequency dissimilarity
        double freqDiss = 0.0;
        for (int j = 0; j < ptsPerFFT; ++j) { freqDiss += fabs(fftAHost[j] - fftBHost[j]); }
        freqDiss *= 0.5;

        diss[i] = timeDiss * freqDiss; // Combine time / frequency dissimilarity
    }

    // Cleanup
    delete[] hil1Host;
    delete[] hil2Host;
    delete[] fftInputHost;
    delete[] envelope1Host;
    delete[] envelope2Host;
    delete[] magnitudeHost;
    delete[] fftAHost;
    delete[] fftBHost;

    // Free device memory
    hipFree(deviceFFTInput);
    hipFree(deviceFFTOutput);
    hipFree(deviceMagnitude);
    hipFree(deviceEnvelope1);
    hipFree(deviceEnvelope2);

    hipfftDestroy(fftPlan);

    return diss;
}

// Free allocated memory for audio samples
void freeAudioData(AudioData& audio) {
    for (int ch = 0; ch < audio.numChannels; ++ch) { delete[] audio.samples[ch]; } // Deallocate each channel
    delete[] audio.samples; // Deallocate top level array
}

// Free allocated memory for extracted features
void freeAudioFeatures(AudioFeatures& features) {
    // Free 1D feature arrays
    delete[] features.segmentDuration;
    delete[] features.SPLrms;
    delete[] features.SPLpk;
    delete[] features.impulsivity;
    delete[] features.dissim;
    delete[] features.peakCount;
    
    for (int i = 0; i < features.autocorrRows; ++i) { delete[] features.autocorr[i]; } // Deallocate each row of autocorr
    delete[] features.autocorr; // Deallocate top layer array

    // Reset pointers
    features.segmentDuration = nullptr;
    features.SPLrms = nullptr;
    features.SPLpk = nullptr;
    features.impulsivity = nullptr;
    features.dissim = nullptr;
    features.peakCount = nullptr;
    features.autocorr = nullptr;
}

__global__ void convertToPressureKernel(const double* samples, double* pressure, int numSamples,
            int numBits, double peakVolts, double refSens) {

    int index = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    if (index >= numSamples) { return; } // Exit if out of bounds

    double samp = samples[index]; // Read sample

    // Right shift align
    if (numBits == 24) { samp = static_cast<double>(static_cast<int>(samp) >> 8); }
    else if (numBits == 32) { samp = static_cast<double>(static_cast<int>(samp) >> 16); }

    // Convert to pressure
    pressure[index] = samp * (peakVolts / static_cast<double>(1 << numBits)) * (1.0 / pow(10.0, refSens / 20.0));
}

void gpuConvertToPressure(const double* hostSamples, double* hostPressure, int length,
            int numBits, double peakVolts, double refSens) {

    // Allocate device memory
    double* deviceSamples;
    double* devicePressure;
    hipMalloc(&deviceSamples, sizeof(double) * length);
    hipMalloc(&devicePressure, sizeof(double) * length);

    hipMemcpy(deviceSamples, hostSamples, sizeof(double) * length, hipMemcpyHostToDevice); // Copy samples to device

    // CUDA launch configuration
    int threadsPerBlock = 256;
    int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    convertToPressureKernel <<<blocks, threadsPerBlock >>> (deviceSamples, devicePressure,
            length, numBits, peakVolts, refSens);

    hipMemcpy(hostPressure, devicePressure, sizeof(double) * length, hipMemcpyDeviceToHost); // Copy pressure to host
    
    // Cleanup
    hipFree(deviceSamples);
    hipFree(devicePressure);
}

// Main feature extraction
AudioFeatures featureExtraction(int numBits, int peakVolts, const fs::path& filePath,
    double refSens, int timewin, double avtime, int fftWin, int calTone, int flow,
    int fhigh, int downsampleFactor, bool omitPartialMinute) {

    string fixedFilePath = fixFilePath(filePath.string()); // Make file path Windows compatible
    AudioData audio = audioRead(filePath.string()); // Read all samples / metadata

    int sampFreq = audio.sampleRate; // Sampling frequency
    int audioSamplesLen = audio.numFrames; // # of audio frames

    double* pressure = new double[audioSamplesLen]; // Buffer for pressure waveform

    double* flatSamples = audio.samples[0]; // First channel
    // Convert to pressure (Pascals)
    gpuConvertToPressure(flatSamples, pressure, audioSamplesLen, numBits, peakVolts, refSens);
    freeAudioData(audio); // Deallocate original audio data

    if (downsampleFactor != -1) { // Downsample
        int newLen = 0;
        double* downsampled = downSample(pressure, audioSamplesLen, downsampleFactor, newLen); // Shortened array
        delete[] pressure; // Deallocate longer array
        pressure = downsampled;
        audioSamplesLen = newLen;
        sampFreq /= downsampleFactor;
    }

    if ((calTone == 1 && audioSamplesLen > 6 * sampFreq)) { // Remove first six seconds if calibration tone is present
        int newLen = audioSamplesLen - 6 * sampFreq;
        double* shifted = new double[newLen]; // Shortened array
        memcpy(shifted, pressure + 6 * sampFreq, sizeof(double) * newLen);
        delete[] pressure; // Deallocate longer array
        pressure = shifted;
        audioSamplesLen = newLen;
    }

    // Only include full minutes of recordings
    if ((omitPartialMinute)) {
        double currentDuration = static_cast<double>(audioSamplesLen) / sampFreq;
        double fullMinutesDuration = floor(currentDuration / 60.0) * 60.0; // Find full minutes
        int croppedFrames = static_cast<int>(sampFreq * fullMinutesDuration); // # of frames for full minutes
        if (croppedFrames < audioSamplesLen) { // Trim if full minutes < total time
            double* trimmed = new double[croppedFrames]; // Allocate space for full minute
            memcpy(trimmed, pressure, sizeof(double) * croppedFrames); // Only include full minutes
            delete[] pressure; // Delete original array
            pressure = trimmed; // Set samples to only include full minutes
            audioSamplesLen = croppedFrames; // Update metadata
        }
    }

    // Apply bandpass filter
    BandpassFilter filt = bandpassFilter(pressure, audioSamplesLen, 1.0 / sampFreq, flow, fhigh);
    delete[] pressure;

    // Segment length (samples) / # of time windows
    int ptsPerTimeWin = timewin * sampFreq;
    int numTimeWin = filt.length / ptsPerTimeWin;
    int remainder = filt.length % ptsPerTimeWin;
    if (remainder > 0) { ++numTimeWin; } // Partial minute

    // Pad filtered signal to match time window
    int paddedLen = numTimeWin * ptsPerTimeWin;
    double* paddedSignal = new double[paddedLen]();
    // Include padding for consistent row / column lengths
    memcpy(paddedSignal, filt.filteredTimeSeries, sizeof(double) * filt.length);
    delete[] filt.filteredTimeSeries;
    filt.filteredTimeSeries = nullptr;

    // Initialize audio features struct
    AudioFeatures features = {};
    features.segmentDurationLen = numTimeWin;
    features.segmentDuration = new int[numTimeWin];

    // Signal to time windows
    double** timechunkMatrix = new double* [numTimeWin];
    for (int i = 0; i < numTimeWin; ++i) {
        timechunkMatrix[i] = &paddedSignal[i * ptsPerTimeWin];
        // Duration per segment
        if (i == numTimeWin - 1 && remainder > 0)
        { features.segmentDuration[i] = static_cast<int>(round(static_cast<double>(remainder) / sampFreq)); }
        else { features.segmentDuration[i] = timewin; }
    }

    // Allocate space for features
    features.SPLrmsLen = features.SPLpkLen = features.impulsivityLen = numTimeWin;
    features.SPLrms = new double[numTimeWin];
    features.SPLpk = new double[numTimeWin];
    features.impulsivity = new double[numTimeWin];

    // Calculate SPLrms, SPLpk, / impulsivity for each time segment
    for (int i = 0; i < numTimeWin; ++i) {
        const double* chunk = timechunkMatrix[i];
        double sumSq = 0.0, peak = 0.0;
        for (int j = 0; j < ptsPerTimeWin; ++j) {
            double temp = chunk[j];
            sumSq += temp * temp;
            if (fabs(temp) > peak) { peak = fabs(temp); }
        }
        double rms = sqrt(sumSq / ptsPerTimeWin);
        features.SPLrms[i] = 20.0 * log10(max(rms, 1e-12));
        features.SPLpk[i] = 20.0 * log10(max(peak, 1e-12));
        features.impulsivity[i] = calculateKurtosis(chunk, ptsPerTimeWin);
    }

    // Calculate autocorr / peakcount
    SoloPer per = calculatePeriodicity(paddedSignal, paddedLen, sampFreq, timewin, avtime);

    features.peakCountLen = numTimeWin;
    features.peakCount = new int[numTimeWin];
    for (int i = 0; i < numTimeWin; ++i) { features.peakCount[i] = per.peakCount[i]; } // Write peakCount to features
    delete[] per.peakCount; // Free original array

    features.autocorrRows = per.autocorrRows;
    features.autocorrCols = per.autocorrCols;
    features.autocorr = new double* [per.autocorrRows];
    for (int i = 0; i < per.autocorrRows; ++i) { features.autocorr[i] = per.autocorr[i]; } // Write autocorr to features matrix
    delete[] per.autocorr; // Free original array

    // Calculate dissim
    int dissimLen = 0;
    features.dissim = calculateDissimGPU(timechunkMatrix, ptsPerTimeWin, numTimeWin, fftWin, sampFreq, dissimLen);
    features.dissimLen = dissimLen;

    // Deallocate temporary arrays
    delete[] timechunkMatrix;
    delete[] paddedSignal;

    return features;
}

// Copy input file name to output file record
tm extractBaseTime(const string& filename) {
    tm baseTime = {}; // Fields initialized to zero
    smatch match; // Will store matched part
    regex pattern1(R"((\d{8})_(\d{6}))"); // Matches YYYYMMDD_HHMMSS
    regex pattern2(R"(.*\.(\d{6})(\d{6}))"); // Matches XXXX.YYMMDDHHMMSS

    // Find date / time from file name
    if (regex_search(filename, match, pattern1) && match.size() == 3) {
        string date = match[1]; // Date
        string time = match[2]; // Time

        baseTime.tm_year = stoi(date.substr(0, 4)) - 1900; // Years since 1900: 4 digits
        baseTime.tm_mon = stoi(date.substr(4, 2)) - 1; // Zero based month
        baseTime.tm_mday = stoi(date.substr(6, 2)); // Day of month
        baseTime.tm_hour = stoi(time.substr(0, 2)) - 1; // Hour
        baseTime.tm_min = stoi(time.substr(2, 2)); // Minute
        baseTime.tm_sec = stoi(time.substr(4, 2)); // Second
    }
    else if (regex_search(filename, match, pattern2) && match.size() == 3) {
        string date = match[1]; // YYMMDD
        string time = match[2]; // HHMMSS

        int year = stoi(date.substr(0, 2)); // Year: 2 digits
        // Assume years 00-40 are 2000s, 41-99 are 1900s
        if (year <= 40) { baseTime.tm_year = year + 100; } // 2000-2040
        else { baseTime.tm_year = year; } // 1941-1999
        baseTime.tm_mon = stoi(date.substr(2, 2)) - 1; // Month
        baseTime.tm_mday = stoi(date.substr(4, 2)); // Day
        baseTime.tm_hour = stoi(time.substr(0, 2)) - 1; // Hour
        baseTime.tm_min = stoi(time.substr(2, 2)); // Minute
        baseTime.tm_sec = stoi(time.substr(4, 2)); // Second
    }

    return baseTime;
}

// Export saved features to CSV file
void saveFeaturesToCSV(const char* filename, const char** filenames, int numFiles, const AudioFeatures* allFeatures) {
    ofstream outputFile(filename);
    if (!outputFile.is_open()) { // Error opening file
        cerr << "Error: Unable to open output file: " << filename << endl;
        return;
    }

    // Determine max autocorr matrix size
    int maxAutocorrRows = 0;
    int maxAutocorrCols = 0;

    for (int i = 0; i < numFiles; ++i) { // Find # of autocorrelation rows/cols for consistent formatting
        const AudioFeatures& feature = allFeatures[i];
        if (feature.autocorr != nullptr && feature.autocorrRows > 0 && feature.autocorrCols > 0) {
            if (feature.autocorrRows > maxAutocorrRows) { maxAutocorrRows = feature.autocorrRows; }
            if (feature.autocorrCols > maxAutocorrCols) { maxAutocorrCols = feature.autocorrCols; }
        }
    }

    // Allocate array for valid autocorr columns - Avoids printing empty columns
    bool* validAutocorrCols = new bool[maxAutocorrCols];
    for (int i = 0; i < maxAutocorrCols; ++i) { validAutocorrCols[i] = false; }

    // Remove extra autocorr columns
    for (int i = 0; i < numFiles; ++i) {
        const AudioFeatures& feature = allFeatures[i];
        if (feature.autocorr != nullptr) {
            for (int row = 0; row < feature.autocorrRows; ++row) {
                for (int col = 0; col < feature.autocorrCols; ++col) {
                    if (!isnan(feature.autocorr[row][col])) { validAutocorrCols[col] = true; } // Ensure no NaN values
                }
            }
        }
    }

    // CSV Header
    outputFile << "Filename,Year,Month,Day,Hour,Minute,SegmentDuration,SPLrms,SPLpk,Impulsivity,Dissimilarity,PeakCount";
    for (int i = 0; i < maxAutocorrCols; ++i) {
        if (validAutocorrCols[i]) { outputFile << ",Autocorr_" << i; }
    }
    outputFile << "\n";

    // Write one row per time segment for each file
    for (int fileIdx = 0; fileIdx < numFiles; ++fileIdx) {
        const AudioFeatures& features = allFeatures[fileIdx];

        // Find max length of features (Most features are length n, dissim is n - 1)
        int maxLength = features.SPLrmsLen;

        // Convert timestamp to time_t
        tm baseTime = extractBaseTime(filenames[fileIdx]);
        time_t baseEpoch = mktime(&baseTime);
        tm* firstTime = localtime(&baseEpoch);

        // Use NaN for empty indices - Only applies to dissim
        bool useNanTimestamp = false;
        if (!firstTime || (firstTime->tm_year + 1900) < 1900) { useNanTimestamp = true; } // Use NaN if no timestamp present

        // Iterate through segments
        for (int i = 0; i < maxLength; ++i) {
            // Calculate timestamp per minute
            time_t currentEpoch = baseEpoch + i * 60; // Use correct indexing for minutes
            tm* currentTime = localtime(&currentEpoch); // Convert from Unix time to time structure

            outputFile << filenames[fileIdx] << ","; // Write filename

            // Write timestamp or NaN
            if (useNanTimestamp || !currentTime) {  outputFile << "NaN,NaN,NaN,NaN,NaN,"; }
            else {
                outputFile << (currentTime->tm_year + 1900) << ","
                           << (currentTime->tm_mon + 1) << ","
                           << currentTime->tm_mday << ","
                           << currentTime->tm_hour << ","
                           << currentTime->tm_min << ",";
            }

            // Segment duration
            if (i < features.segmentDurationLen) { outputFile << features.segmentDuration[i]; }
            else { outputFile << "NaN"; }
            outputFile << ",";

            // SPLrms
            if (i < features.SPLrmsLen) outputFile << features.SPLrms[i];
            else { outputFile << "NaN"; }
            outputFile << ",";

            // SPLpk
            if (i < features.SPLpkLen) outputFile << features.SPLpk[i];
            else { outputFile << "NaN"; }
            outputFile << ",";

            // Impulsivity
            if (i < features.impulsivityLen) outputFile << features.impulsivity[i];
            else { outputFile << "NaN"; }
            outputFile << ",";

            // Dissim
            if (i < features.dissimLen) outputFile << features.dissim[i];
            else { outputFile << "NaN"; }
            outputFile << ",";

            // Peakcount
            if (i < features.peakCountLen) outputFile << features.peakCount[i];
            else { outputFile << "NaN"; }
            
            // Autocorr
            for (int j = 0; j < maxAutocorrCols; ++j) {
                if (validAutocorrCols[j]) {
                    outputFile << ",";
                    if (features.autocorr && i < features.autocorrRows && j < features.autocorrCols)
                        { outputFile << features.autocorr[i][j]; }
                    else { outputFile << "NaN"; }
                }
            }

            outputFile << "\n"; // End of row
        }
    }

    // Clean up
    delete[] validAutocorrCols;
    outputFile.close();
}

// Sort input files
void bubbleSort(char arr[][128], int n) {
    char temp[128]; // Buffer for swapping strings
    for (int i = 0; i < n - 1; ++i) { // Iterate through array
        for (int j = 0; j < n - i - 1; ++j) { // Compare adjacent elements up to unsorted portion
            if (strcmp(arr[j], arr[j + 1]) > 0) { // Lexicographically compare two strings
                // Swap strings using temp buffer
                strcpy(temp, arr[j]);
                strcpy(arr[j], arr[j + 1]);
                strcpy(arr[j + 1], temp);
            }
        }
    }
}

void threadWrapper(ThreadArgs& args) {
    try { // Use parallel processing to extract features from multiple files at a time
        while (true) { // Find next index
            int index = args.nextIndex->fetch_add(1);
            if (index >= args.totalFiles) { break; } // End of input files

            // Display current file being processed
            cout << "Processing file index " << index << ": " << args.filePaths[index] << "\n";
            cerr.flush();

            fs::path filePath(args.filePaths[index]);

            // Extract base time from filename
            string filename_str = filePath.filename().string();
            tm extractedTime = extractBaseTime(filename_str);

            // Store the extracted time information
            args.fileTimeInfo[index].baseTime = extractedTime;
            args.fileTimeInfo[index].filename = filename_str;

            // Check if time was successfully extracted
            bool timeValid = (extractedTime.tm_year > 0 || extractedTime.tm_mon >= 0 ||
                extractedTime.tm_mday > 0 || extractedTime.tm_hour >= 0);
            args.fileTimeInfo[index].timeExtracted = timeValid;

            // Perform feature extraction for current input file
            AudioFeatures features = featureExtraction(args.numBits, args.peakVolts,
                filePath, args.RS, args.timeWin, args.avTime, args.fftWin,
                args.arti, args.fLow, args.fHigh, args.downSample, args.omitPartialMinute);

            args.allFeatures[index] = features;

            // Display base filename in output results
            strcpy(args.filenames[index], filename_str.c_str()); // Copy 
            args.filenames[index][511] = '\0'; // Null terminate for safe handling
        }
    }
    // Display any errors
    catch (const exception& e) { cerr << "Exception in thread: " << e.what() << "\n"; }
    catch (...) { cerr << "Unknown exception in thread\n"; }
}

// Process directory of sound files with user-given parameters
int main(int argc, char* argv[]) {
    using namespace std; // Standard namespace
    using namespace chrono; // Time tracking

    auto start = high_resolution_clock::now(); // Starting time to show runtime performance

    // Use first available GPU
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < 1) {
        fprintf(stderr, "No CUDA devices available.\n");
        exit(EXIT_FAILURE);
    }
    int deviceID = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceID);
    hipSetDevice(deviceID);

    // Default arguments if unspecified
    char inputDir[512] = {}, outputFile[512] = {};
    int numBits = 16, peakVolts = 2, arti = 1, timeWin = 60, fftWin = 1,
        fLow = 1, fHigh = 192000, maxThreads = 4, downSample = -1;
    double RS = -178.3, avTime = 0.1;
    bool omitPartialMinute = false;

    // Command line index parsing
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--omit_partial_minute") == 0) { omitPartialMinute = true; }
        else if (strcmp(argv[i], "--input") == 0 && i + 1 < argc) { strcpy(inputDir, argv[++i]); }
        else if (strcmp(argv[i], "--output") == 0 && i + 1 < argc) { strcpy(outputFile, argv[++i]); }
        else if (strcmp(argv[i], "--num_bits") == 0) { numBits = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--RS") == 0) { RS = atof(argv[++i]); }
        else if (strcmp(argv[i], "--peak_volts") == 0) { peakVolts = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--arti") == 0) { arti = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--timewin") == 0) { timeWin = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--fft_win") == 0) { fftWin = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--avtime") == 0) { avTime = atof(argv[++i]); }
        else if (strcmp(argv[i], "--flow") == 0) { fLow = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--fhigh") == 0) { fHigh = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--max_threads") == 0) { maxThreads = atoi(argv[++i]); }
        else if (strcmp(argv[i], "--downsample") == 0) { downSample = atoi(argv[++i]); }
    }

    // Count .wav files
    int totalFiles = 0;
    for (const auto& entry : fs::directory_iterator(inputDir)) {
        if (entry.path().extension() == ".wav") { ++totalFiles; } // Only read .wav files
    }

    if (totalFiles == 0) { // No .wav files in directory
        cerr << "No valid .wav files found in " << inputDir << "\n";
        return 1;
    }

    // Allocate arrays large enough for all files
    char (*filePaths)[128] = new char[totalFiles][128]; // Path to files
    char (*filenames)[128] = new char[totalFiles][128]; // Base filenames
    AudioFeatures* allFeatures = new AudioFeatures[totalFiles]; // Initialize AudioFeatures struct
    FileTimeInfo* fileTimeInfo = new FileTimeInfo[totalFiles]; // Time info for each file

    // Fill filePaths with file names
    int index = 0;
    for (const auto& entry : fs::directory_iterator(inputDir)) {
        if (entry.path().extension() == ".wav") {
            strcpy(filePaths[index], entry.path().string().c_str());
            filePaths[index][511] = '\0'; // Reserve final index for terminal character
            ++index;
        }
    }

    bubbleSort(filePaths, totalFiles); // Sort file paths alphabetically for in-order data processing

    // Thread setup
    atomic<int> nextIndex(0);
    int availableThreads = max(1, thread::hardware_concurrency());
    int numThreads = min(maxThreads, availableThreads);

    // Thread arguments
    ThreadArgs args;
    args.nextIndex = &nextIndex;
    args.totalFiles = totalFiles;
    args.filePaths = filePaths;
    args.filenames = filenames;
    args.allFeatures = allFeatures;
    args.fileTimeInfo = fileTimeInfo;
    args.numBits = numBits;
    args.peakVolts = peakVolts;
    args.RS = RS;
    args.timeWin = timeWin;
    args.avTime = avTime;
    args.fftWin = fftWin;
    args.arti = arti;
    args.fLow = fLow;
    args.fHigh = fHigh;
    args.downSample = downSample;
    args.omitPartialMinute = omitPartialMinute;

    // Launch threads
    thread* threads = new thread[numThreads];
    for (int i = 0; i < numThreads; ++i) { threads[i] = thread(threadWrapper, ref(args)); }
    for (int i = 0; i < numThreads; ++i) { threads[i].join(); } // Synchronize threads
    delete[] threads;

    const char** fileNames = new const char* [totalFiles];
    for (int i = 0; i < totalFiles; ++i) { fileNames[i] = filenames[i]; } // Convert to const char* array

    // Save calculated features to output
    saveFeaturesToCSV(outputFile, fileNames, totalFiles, allFeatures);
    cout << "Saved features for " << totalFiles << " files to " << outputFile << "\n";

    for (int i = 0; i < totalFiles; ++i) { freeAudioFeatures(allFeatures[i]); } // Free features for each file

    // Cleanup
    delete[] filePaths;
    delete[] filenames;
    delete[] allFeatures;
    delete[] fileTimeInfo;
    delete[] fileNames;
    fftw_cleanup(); // Clean up internal memory

    auto stop = high_resolution_clock::now(); // Ending time to show runtime performance
    duration<double> elapsed = duration_cast<duration<double>>(stop - start);
    cout << "Runtime: " << elapsed.count() << " seconds\n"; // Total execution time

    return 0;
}
